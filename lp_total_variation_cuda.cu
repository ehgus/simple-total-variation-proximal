#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <cmath>

// Device function: spatial_diff_T operation
// Input: z_data (input array ndims, total_elements), dims, strides, ndims
// Output: returns the result
template<typename T>
__device__ T spatial_diff_T_device(const T* z_data, int idx, int ndims, const int* dims, const int* strides, const int total_elements) {
    // Calculate multi-dimensional coordinates from linear index
    int temp_idx = idx;
    int coords[4];
    for (int d = 0; d < ndims; d++) {
        coords[d] = temp_idx % dims[d];
        temp_idx /= dims[d];
    }

    // Initialize with sum across last dimension (sum of z over last dim)
    T sum_val = T(0.0);
    for (int dim = 0; dim < ndims; dim++) {
        sum_val += z_data[idx + total_elements * dim];
    }

    // Compute transpose spatial differences for each dimension
    for (int dim = 0; dim < ndims; dim++) {
        // Apply forward circular shift: move forward by 1 in current dimension
        int neighbor_coord = (coords[dim] + 1) % dims[dim];

        // Calculate neighbor index efficiently using pre-computed strides
        int neighbor_idx = idx + (neighbor_coord - coords[dim]) * strides[dim];

        // Subtract the shifted value (transpose operation)
        sum_val -= z_data[neighbor_idx + total_elements * dim];
    }

    // Return result
    return sum_val;
}

// Device function: spatial_diff operation
// Input: x_tmp (total_elements), dims, strides, ndims
// Output: ndims values for position idx (stored in result array)
template<typename T>
__device__ void spatial_diff_device(const T* x_tmp_data, T* result, int idx, int ndims, const int* dims, const int* strides) {
    const T center_data = x_tmp_data[idx];

    // Calculate multi-dimensional coordinates from linear index
    int temp_idx = idx;
    int coords[4];
    for (int d = 0; d < ndims; d++) {
        coords[d] = temp_idx % dims[d];
        temp_idx /= dims[d];
    }

    // Compute spatial differences for each dimension
    for (int dim = 0; dim < ndims; dim++) {
        // Apply circular shift: move backward by 1 in current dimension
        int neighbor_coord = (coords[dim] - 1 + dims[dim]) % dims[dim];

        // Calculate neighbor index efficiently using pre-computed strides
        int neighbor_idx = idx + (neighbor_coord - coords[dim]) * strides[dim];

        // Compute difference: current - shifted
        result[dim] = center_data - x_tmp_data[neighbor_idx];
    }
}

// Device function: L2 norm projection
// Input: z_element, z_tmp_element (ndims values each)
// Output: projected z_element (ndims values)
template<typename T>
__device__ void l2_norm_projection_device(T* z_element, const T* z_tmp_element, int ndims) {
    // Calculate the L2 norm of z_tmp_element
    T norm_val = T(0.0);
    for (int d = 0; d < ndims; d++) {
        norm_val += z_tmp_element[d] * z_tmp_element[d];
    }
    norm_val = sqrt(norm_val);

    // Apply L2 projection: z = z_tmp / max(1, ||z_tmp||_2)
    T scale_factor = (norm_val > T(1.0)) ? norm_val : T(1.0);
    for (int d = 0; d < ndims; d++) {
        z_element[d] = z_tmp_element[d] / scale_factor;
    }
}

// Device function: L-infinity norm projection
// Input: z_element, z_tmp_element (ndims values each)
// Output: projected z_element (ndims values)
template<typename T>
__device__ void linf_norm_projection_device(T* z_element, const T* z_tmp_element, int ndims) {
    // L-infinity projection: clamp each component to [-1, 1]
    // y = max(min(x, 1), -1)
    for (int d = 0; d < ndims; d++) {
        T val = z_tmp_element[d];
        if (val > T(1.0)) {
            z_element[d] = T(1.0);
        } else if (val < T(-1.0)) {
            z_element[d] = T(-1.0);
        } else {
            z_element[d] = val;
        }
    }
}

// Device function: Generic projection dispatcher
// Input: z_element, z_tmp_element (ndims values each), projection_type
// Output: projected z_element (ndims values)
template<typename T>
__device__ void unit_ball_projection_device(T* z_element, const T* z_tmp_element, int ndims, int projection_type) {
    if (projection_type == 2) {
        // L2 projection
        l2_norm_projection_device(z_element, z_tmp_element, ndims);
    } else if (projection_type == -1) {
        // L-infinity projection (use -1 to represent infinity)
        linf_norm_projection_device(z_element, z_tmp_element, ndims);
    } else {
        // Default to L2 projection
        l2_norm_projection_device(z_element, z_tmp_element, ndims);
    }
}

// Kernel 1: spatial_diff_T operation
template<typename T>
__global__ void spatial_diff_T_kernel(
    T* x_tmp_data,          // Output: x_tmp array (total_elements)
    const T* x_data,        // Input: x (total_elements)
    const T* z_data,        // Input: z (ndims, total_elements)
    T w,                    // Weight parameter
    int ndims,              // Number of dimensions
    const int* dims,        // Dimension sizes
    const int* strides,     // Pre-computed strides
    int total_elements      // Total number of elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= total_elements) return;

    // Step 1: x_tmp = x + w * spatial_diff_T(z)
    T spatial_diff_T_result = spatial_diff_T_device(z_data, idx, ndims, dims, strides, total_elements);
    x_tmp_data[idx] = x_data[idx] + w * spatial_diff_T_result;
}

// Kernel 2: spatial_diff operation and z update
template<typename T>
__global__ void spatial_diff_z_update_kernel(
    T* z_data,              // Input/Output: z (ndims, total_elements)
    const T* x_tmp_data,    // Input: x_tmp array (total_elements)
    T v,                    // Norm weight parameter
    int projection_type,    // Projection type: 2 for L2, -1 for L-infinity
    int ndims,              // Number of dimensions
    const int* dims,        // Dimension sizes
    const int* strides,     // Pre-computed strides
    int total_elements      // Total number of elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= total_elements) return;

    // Local arrays for this thread's computation
    T z_tmp_local[4];  // Support up to 4 dimensions
    T spatial_diff_result[4];

    // Step 2: z_tmp = z - v * spatial_diff(x_tmp)
    spatial_diff_device(x_tmp_data, spatial_diff_result, idx, ndims, dims, strides);
    for (int d = 0; d < ndims; d++) {
        z_tmp_local[d] = z_data[idx + total_elements * d] - v * spatial_diff_result[d];
    }

    // Step 3: z = v * norm.projection(z, z_tmp / v)
    T z_tmp_scaled[4];
    for (int d = 0; d < ndims; d++) {
        z_tmp_scaled[d] = z_tmp_local[d] / v;
    }

    T z_projected[4];
    unit_ball_projection_device(z_projected, z_tmp_scaled, ndims, projection_type);

    for (int d = 0; d < ndims; d++) {
        z_data[idx + total_elements * d] = v * z_projected[d];
    }
}

// Template function for the optimization algorithm
template<typename T>
void lp_total_variation_optimize(
    const T* x_data,            // Input data
    T* y_data,                  // Output data
    T w, T v,                   // Weight parameters
    int niter,                  // Number of iterations
    int projection_type,        // Projection type
    int ndims_x,                // Number of dimensions
    int total_elements,         // Total elements
    int z_total_elements,       // Total z elements
    const int* d_dims,          // Device dimensions
    const int* d_strides        // Device strides
) {
    // Calculate optimal block sizes
    int threadsPerBlock_sdiff_T, minblocksPerGrid_sdiff_T, blocksPerGrid_sdiff_T;
    int threadsPerBlock_diff_z, minblocksPerGrid_diff_z, blocksPerGrid_diff_z;

    hipOccupancyMaxPotentialBlockSize(&minblocksPerGrid_sdiff_T, &threadsPerBlock_sdiff_T,
        (void*)spatial_diff_T_kernel<T>, 0, total_elements);
    blocksPerGrid_sdiff_T = (total_elements + threadsPerBlock_sdiff_T - 1) / threadsPerBlock_sdiff_T;

    hipOccupancyMaxPotentialBlockSize(&minblocksPerGrid_diff_z, &threadsPerBlock_diff_z,
        (void*)spatial_diff_z_update_kernel<T>, 0, total_elements);
    blocksPerGrid_diff_z = (total_elements + threadsPerBlock_diff_z - 1) / threadsPerBlock_diff_z;

    // Allocate native CUDA memory for z array (optimization variable)
    T *z_data;
    hipMalloc(&z_data, z_total_elements * sizeof(T));
    hipMemset(z_data, 0, z_total_elements * sizeof(T)); // Initialize to zero

    // Main optimization loop
    for (int iter = 0; iter < niter; iter++) {
        // Step 1: spatial_diff(y) and z update
        spatial_diff_z_update_kernel<T><<<blocksPerGrid_diff_z, threadsPerBlock_diff_z>>>(
            z_data, y_data, v, projection_type, ndims_x, d_dims, d_strides, total_elements);

        // Step 2: y = x + w * spatial_diff_T(z)
        spatial_diff_T_kernel<T><<<blocksPerGrid_sdiff_T, threadsPerBlock_sdiff_T>>>(
            y_data, x_data, z_data, w, ndims_x, d_dims, d_strides, total_elements);
    }

    // Wait for completion
    hipDeviceSynchronize();

    // Check for kernel errors
    if (hipGetLastError() != hipSuccess) {
        mexErrMsgTxt("CUDA kernel execution failed");
    }

    // Clean up native CUDA arrays
    hipFree(z_data);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    // Initialize GPU
    mxInitGPU();

    // Input validation
    if (nrhs != 5) {
        mexErrMsgTxt("Five inputs required: x, w, v, niter, p_norm");
    }
    if (nlhs > 1) {
        mexErrMsgTxt("Too many output arguments");
    }

    // Get input arrays and parameters
    const mxGPUArray *x_gpu = mxGPUCreateFromMxArray(prhs[0]);
    double w = mxGetScalar(prhs[1]);
    double v = mxGetScalar(prhs[2]);
    int niter = (int)mxGetScalar(prhs[3]);
    double p_norm = mxGetScalar(prhs[4]);

    // Determine projection type from p_norm
    // Following MATLAB logic: obj.norm = LpUnitBall(round(1/(1-1/p)))
    int projection_type;
    if (isinf(p_norm)) {
        projection_type = -1;  // L-infinity projection
    } else {
        double dual_p = 1.0 / (1.0 - 1.0 / p_norm);
        int rounded_dual_p = (int)round(dual_p);
        if (rounded_dual_p == 2) {
            projection_type = 2;   // L2 projection
        } else if (isinf(dual_p)) {
            projection_type = -1;  // L-infinity projection
        } else {
            mexErrMsgTxt("p value other than 0 and Inf is not supported");
            projection_type = 2;   // Default to L2 projection
        }
    }

    // Validate data types
    mxClassID x_class = mxGPUGetClassID(x_gpu);
    if (x_class != mxDOUBLE_CLASS && x_class != mxSINGLE_CLASS) {
        mexErrMsgTxt("Input array must be of type double or single");
    }

    // Get dimensions of input x
    const mwSize *x_dims = mxGPUGetDimensions(x_gpu);
    int ndims_x = mxGPUGetNumberOfDimensions(x_gpu);

    if (ndims_x > 4) {
        mexErrMsgTxt("Maximum 4 dimensions supported");
    }

    // Calculate total elements
    int total_elements = 1;
    for (int i = 0; i < ndims_x; i++) {
        total_elements *= x_dims[i];
    }

    // Calculate z array size: total_elements * ndims_x
    int z_total_elements = total_elements * ndims_x;

    // Create output array y for final result (same size as input x)
    mxGPUArray *y_result = mxGPUCopyFromMxArray(prhs[0]);

    // Copy dimensions and compute strides
    int *d_dims, *d_strides;
    int dims_host[4], strides_host[4];

    for (int i = 0; i < ndims_x; i++) {
        dims_host[i] = (int)x_dims[i];
    }

    // Compute strides
    strides_host[0] = 1;
    for (int d = 1; d < ndims_x; d++) {
        strides_host[d] = strides_host[d-1] * dims_host[d-1];
    }

    hipMalloc(&d_dims, ndims_x * sizeof(int));
    hipMalloc(&d_strides, ndims_x * sizeof(int));
    hipMemcpy(d_dims, dims_host, ndims_x * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_strides, strides_host, ndims_x * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel based on data type
    if (x_class == mxDOUBLE_CLASS) {
        // Get pointers to data
        const double *x_data = (const double*)mxGPUGetDataReadOnly(x_gpu);
        double *y_data = (double*)mxGPUGetData(y_result);

        // Run optimization algorithm
        lp_total_variation_optimize<double>(
            x_data, y_data, (double)w, (double)v, niter, projection_type,
            ndims_x, total_elements, z_total_elements, d_dims, d_strides);
    } else { // mxSINGLE_CLASS
        // Get pointers to data
        const float *x_data = (const float*)mxGPUGetDataReadOnly(x_gpu);
        float *y_data = (float*)mxGPUGetData(y_result);

        // Run optimization algorithm
        lp_total_variation_optimize<float>(
            x_data, y_data, (float)w, (float)v, niter, projection_type,
            ndims_x, total_elements, z_total_elements, d_dims, d_strides);
    }

    // Clean up device memory
    hipFree(d_dims);
    hipFree(d_strides);

    // Create output
    plhs[0] = mxGPUCreateMxArrayOnGPU(y_result);

    // Clean up
    mxGPUDestroyGPUArray(x_gpu);
    mxGPUDestroyGPUArray(y_result);
}